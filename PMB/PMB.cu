#include "hip/hip_runtime.h"
#include <_Time.h>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>
//1024: {16, 16, 64}
//2048: {16, 16, 64}

unsigned int edgeTable[256]
{
	0x0  , 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c,
	0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
	0x190, 0x99 , 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c,
	0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
	0x230, 0x339, 0x33 , 0x13a, 0x636, 0x73f, 0x435, 0x53c,
	0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
	0x3a0, 0x2a9, 0x1a3, 0xaa , 0x7a6, 0x6af, 0x5a5, 0x4ac,
	0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
	0x460, 0x569, 0x663, 0x76a, 0x66 , 0x16f, 0x265, 0x36c,
	0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
	0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0xff , 0x3f5, 0x2fc,
	0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
	0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x55 , 0x15c,
	0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
	0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0xcc ,
	0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
	0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc,
	0xcc , 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
	0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c,
	0x15c, 0x55 , 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
	0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc,
	0x2fc, 0x3f5, 0xff , 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
	0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c,
	0x36c, 0x265, 0x16f, 0x66 , 0x76a, 0x663, 0x569, 0x460,
	0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac,
	0x4ac, 0x5a5, 0x6af, 0x7a6, 0xaa , 0x1a3, 0x2a9, 0x3a0,
	0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c,
	0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x33 , 0x339, 0x230,
	0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c,
	0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x99 , 0x190,
	0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c,
	0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x0
};

unsigned short distinctEdgesTable[256]
{
	0x0000, 0xE001, 0x8001, 0x6002, 0x0001, 0xE002, 0x8002, 0x6003,
	0x4001, 0xA002, 0xC002, 0x2003, 0x4002, 0xA003, 0xC003, 0x2002,
	0x2001, 0xC002, 0xA002, 0x4003, 0x2002, 0xC003, 0xA003, 0x4004,
	0x6002, 0x8003, 0xE003, 0x0004, 0x6003, 0x8004, 0xE004, 0x0003,
	0x0001, 0xE002, 0x8002, 0x6003, 0x0002, 0xE003, 0x8003, 0x6004,
	0x4002, 0xA003, 0xC003, 0x2004, 0x4003, 0xA004, 0xC004, 0x2003,
	0x2002, 0xC003, 0xA003, 0x4002, 0x2003, 0xC004, 0xA004, 0x4003,
	0x6003, 0x8004, 0xE004, 0x0003, 0x6004, 0x8005, 0xE005, 0x0002,
	0x0001, 0xE002, 0x8002, 0x6003, 0x0002, 0xE003, 0x8003, 0x6004,
	0x4002, 0xA003, 0xC003, 0x2004, 0x4003, 0xA004, 0xC004, 0x2003,
	0x2002, 0xC003, 0xA003, 0x4004, 0x2003, 0xC004, 0xA004, 0x4005,
	0x6003, 0x8004, 0xE004, 0x0005, 0x6004, 0x8005, 0xE005, 0x0004,
	0x0002, 0xE003, 0x8003, 0x6004, 0x0003, 0xE004, 0x8002, 0x6003,
	0x4003, 0xA004, 0xC004, 0x2005, 0x4004, 0xA005, 0xC003, 0x2002,
	0x2003, 0xC004, 0xA004, 0x4003, 0x2004, 0xC005, 0xA003, 0x4002,
	0x6004, 0x8005, 0xE005, 0x0004, 0x6005, 0x8002, 0xE004, 0x0001,
	0x0001, 0xE002, 0x8002, 0x6003, 0x0002, 0xE003, 0x8003, 0x6004,
	0x4002, 0xA003, 0xC003, 0x2004, 0x4003, 0xA004, 0xC004, 0x2003,
	0x2002, 0xC003, 0xA003, 0x4004, 0x2003, 0xC004, 0xA004, 0x4005,
	0x6003, 0x8002, 0xE004, 0x0003, 0x6004, 0x8003, 0xE005, 0x0002,
	0x0002, 0xE003, 0x8003, 0x6004, 0x0003, 0xE004, 0x8004, 0x6005,
	0x4003, 0xA004, 0xC004, 0x2005, 0x4004, 0xA005, 0xC005, 0x2004,
	0x2003, 0xC004, 0xA004, 0x4003, 0x2004, 0xC005, 0xA005, 0x4004,
	0x6004, 0x8003, 0xE005, 0x0002, 0x6005, 0x8004, 0xE002, 0x0001,
	0x0002, 0xE003, 0x8003, 0x6004, 0x0003, 0xE004, 0x8004, 0x6005,
	0x4003, 0xA004, 0xC004, 0x2005, 0x4002, 0xA003, 0xC003, 0x2002,
	0x2003, 0xC004, 0xA004, 0x4005, 0x2004, 0xC005, 0xA005, 0x4002,
	0x6004, 0x8003, 0xE005, 0x0004, 0x6003, 0x8002, 0xE004, 0x0001,
	0x0003, 0xE004, 0x8004, 0x6005, 0x0004, 0xE005, 0x8003, 0x6004,
	0x4004, 0xA005, 0xC005, 0x2002, 0x4003, 0xA004, 0xC002, 0x2001,
	0x2002, 0xC003, 0xA003, 0x4002, 0x2003, 0xC004, 0xA002, 0x4001,
	0x6003, 0x8002, 0xE004, 0x0001, 0x6002, 0x8001, 0xE001, 0x0000
};

int triTable[256][16]
{
	{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
	{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
	{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
	{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
	{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
	{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
	{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
	{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
	{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
	{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
	{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
	{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
	{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
	{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
	{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
	{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
	{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
	{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
	{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
	{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
	{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
	{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
	{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
	{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
	{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
	{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
	{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
	{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
	{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
	{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
	{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
	{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
	{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
	{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
	{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
	{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
	{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
	{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
	{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
	{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
	{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
	{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
	{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
	{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
	{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
	{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
	{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
	{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
	{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
	{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
	{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
	{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
	{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
	{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
	{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
	{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
	{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
	{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
	{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
	{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
	{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
	{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
	{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
	{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
	{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
	{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
	{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
	{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
	{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
	{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
	{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
	{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
	{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
	{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
	{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
	{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
	{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
	{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
	{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
	{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
	{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
	{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
	{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
	{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
	{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
	{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
	{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
	{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
	{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
	{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
	{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
	{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
	{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
	{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
	{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
	{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
	{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
	{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
	{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
	{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
	{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
	{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
	{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
	{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
	{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
	{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
	{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
	{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
	{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
	{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
	{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
	{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
	{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
	{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
	{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
	{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
	{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
	{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
	{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
	{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
	{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
	{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
	{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
	{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
	{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
	{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
	{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
	{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
	{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
	{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
	{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
	{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
	{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
	{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
	{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
	{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
	{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
	{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
	{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
	{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
	{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
	{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
	{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
	{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
	{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
	{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
	{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
	{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
	{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
	{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
	{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
	{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
	{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
	{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
	{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
	{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
	{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
	{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
	{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
	{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
	{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
	{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
	{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
	{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
	{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}
};

uchar4 edgeIDTable[12]
{
	{0, 0, 0, 0},
	{1, 0, 0, 1},
	{0, 1, 0, 0},
	{0, 0, 0, 1},
	{0, 0, 1, 0},
	{1, 0, 1, 1},
	{0, 1, 1, 0},
	{0, 0, 1, 1},
	{0, 0, 0, 2},
	{1, 0, 0, 2},
	{1, 1, 0, 2},
	{0, 1, 0, 2}
};

constexpr unsigned int N(1024);
constexpr unsigned int Nd2(N / 2);
constexpr unsigned int voxelXLv1(16);
constexpr unsigned int voxelYLv1(16);
constexpr unsigned int voxelZLv1(64);
constexpr unsigned int gridXLv1((N - 1) / (voxelXLv1 - 1));
constexpr unsigned int gridYLv1((N - 1) / (voxelYLv1 - 1));
constexpr unsigned int gridZLv1((N - 1) / (voxelZLv1 - 1));
const dim3 BlockSizeLv1{ voxelXLv1, voxelYLv1, 1 };
const dim3 GridSizeLv1{ gridXLv1, gridYLv1, gridZLv1 };
constexpr unsigned int blockNum(gridXLv1* gridYLv1* gridZLv1);

constexpr unsigned int countingThreadNumLv1(128);
constexpr unsigned int countingBlockNumLv1(blockNum / countingThreadNumLv1);

constexpr unsigned int voxelXLv2(4);
constexpr unsigned int voxelYLv2(4);
constexpr unsigned int voxelZLv2(8);
constexpr unsigned int blockXLv2(5);
constexpr unsigned int blockYLv2(5);
constexpr unsigned int blockZLv2(9);
const dim3 BlockSizeLv2{ voxelXLv2 * voxelYLv2, blockXLv2 * blockYLv2, 1 };
constexpr unsigned int voxelNumLv2(blockXLv2* blockYLv2* blockZLv2);

constexpr unsigned int countingThreadNumLv2(1024);
constexpr unsigned int gridXLv2(gridXLv1* blockXLv2);
constexpr unsigned int gridYLv2(gridYLv1* blockYLv2);
constexpr unsigned int gridZLv2(gridZLv1* blockZLv2);

const dim3 BlockSizeGenerating{ voxelXLv2, voxelYLv2, voxelZLv2 };

//A implementation of Parallel Marching Blocks algorithm
__inline__ __device__ float f(unsigned int x, unsigned int y, unsigned int z)
{
	constexpr float d(2.0f / N);
	float xf((int(x - Nd2)) * d);//[-1, 1)
	float yf((int(z - Nd2)) * d);
	float zf((int(z - Nd2)) * d);
	//if (x)return 1;
	//else return -1;
	return 1.f - 16.f * xf * yf * zf - 4.f * (xf * xf + yf * yf + zf * zf);
	//return xf * xf + yf * yf + zf * zf - 1.0f;
}

__inline__ __device__ float zeroPoint(unsigned int x, float v0, float v1)
{
	constexpr float d(2.0f / N);
	return ((x * v1 - (x + 1) * v0) / (v1 - v0) - Nd2) * d;
}

__global__ void computeMinMaxLv1(/*float* data, */float* minMax)
{
	unsigned int laneid;
	asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
	constexpr unsigned int threadNum(voxelXLv1 * voxelYLv1);
	constexpr unsigned int warpNum(threadNum / 32);
	__shared__ float sminMax[64];
	unsigned int x(blockIdx.x * (voxelXLv1 - 1) + threadIdx.x);
	unsigned int y(blockIdx.y * (voxelYLv1 - 1) + threadIdx.y);
	unsigned int z(blockIdx.z * (voxelZLv1 - 1));
	unsigned int tid(threadIdx.x + voxelXLv1 * threadIdx.y);
	unsigned int blockid(blockIdx.x + gridXLv1 * (blockIdx.y + gridYLv1 * blockIdx.z));
	unsigned int warpid(tid >> 5);
	//float v(data[tid + threadNum * blockIdx.x]);
	float v(f(x, y, z));
	float minV(v), maxV(v);
	for (int c0(1); c0 < voxelZLv1; ++c0)
	{
		v = f(x, y, z + c0);
		if (v < minV)minV = v;
		if (v > maxV)maxV = v;
	}
#pragma unroll
	for (int c0(16); c0 > 0; c0 /= 2)
	{
		float t0, t1;
		t0 = __shfl_down_sync(0xffffffffu, minV, c0);
		t1 = __shfl_down_sync(0xffffffffu, maxV, c0);
		if (t0 < minV)minV = t0;
		if (t1 > maxV)maxV = t1;
	}
	if (laneid == 0)
	{
		sminMax[warpid] = minV;
		sminMax[warpid + warpNum] = maxV;
	}
	__syncthreads();
	if (warpid == 0)
	{
		minV = sminMax[laneid];
		maxV = sminMax[laneid + warpNum];
#pragma unroll
		for (int c0(warpNum / 2); c0 > 0; c0 /= 2)
		{
			float t0, t1;
			t0 = __shfl_down_sync(0xffffffffu, minV, c0);
			t1 = __shfl_down_sync(0xffffffffu, maxV, c0);
			if (t0 < minV)minV = t0;
			if (t1 > maxV)maxV = t1;
		}
		if (laneid == 0)
		{
			minMax[blockid * 2] = minV;
			minMax[blockid * 2 + 1] = maxV;
		}
	}
}

__global__ void compatingLv1(float isoValue, float* minMax, unsigned int* blockIndices, unsigned int* countedBlockNum)
{
	unsigned int laneid;
	asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
	constexpr unsigned int warpNum(countingThreadNumLv1 / 32);
	__shared__ unsigned int sums[32];
	unsigned int tid(threadIdx.x);
	unsigned int bIdx(blockIdx.x * countingThreadNumLv1 + tid);
	unsigned int warpid(tid >> 5);
	unsigned int test;
	if (minMax[2 * bIdx] <= isoValue && minMax[2 * bIdx + 1] >= isoValue)test = 1;
	else test = 0;
	unsigned int testSum(test);
#pragma unroll
	for (int c0(1); c0 < 32; c0 *= 2)
	{
		unsigned int tp(__shfl_up_sync(0xffffffffu, testSum, c0));
		if (laneid >= c0)testSum += tp;
	}
	if (laneid == 31)sums[warpid] = testSum;
	__syncthreads();
	if (warpid == 0)
	{
		unsigned warpSum = sums[laneid];
#pragma unroll
		for (int c0(1); c0 < warpNum; c0 *= 2)
		{
			unsigned int tp(__shfl_up_sync(0xffffffffu, warpSum, c0));
			if (laneid >= c0)warpSum += tp;
		}
		sums[laneid] = warpSum;
	}
	__syncthreads();
	if (warpid != 0)testSum += sums[warpid - 1];
	if (tid == countingThreadNumLv1 - 1 && testSum != 0)
		sums[31] = atomicAdd(countedBlockNum, testSum);
	__syncthreads();
	if (test)blockIndices[testSum + sums[31] - 1] = bIdx;
}

__global__ void computeMinMaxLv2(unsigned int* blockIndicesLv1, float* minMax)
{
	unsigned int tid(threadIdx.x);
	unsigned int voxelOffset(threadIdx.y);
	unsigned int blockIndex(blockIndicesLv1[blockIdx.x]);
	unsigned int tp(blockIndex);
	unsigned int x((blockIndex % gridXLv1) * (voxelXLv1 - 1) + (voxelOffset % 5) * (voxelXLv2 - 1) + (tid & 3));
	tp /= gridXLv1;
	unsigned int y((tp % gridYLv1) * (voxelYLv1 - 1) + (voxelOffset / 5) * (voxelYLv2 - 1) + (tid >> 2));
	tp /= gridYLv1;
	unsigned int z(tp * (voxelZLv1 - 1));
	float v(f(x, y, z));
	float minV(v), maxV(v);
	unsigned int idx(2 * (voxelOffset + voxelNumLv2 * blockIdx.x));
	for (int c0(0); c0 < blockZLv2; ++c0)
	{
		for (int c1(1); c1 < voxelZLv2; ++c1)
		{
			v = f(x, y, z + c1);
			if (v < minV)minV = v;
			if (v > maxV)maxV = v;
		}
		z += voxelZLv2 - 1;
#pragma unroll
		for (int c1(8); c1 > 0; c1 /= 2)
		{
			float t0, t1;
			t0 = __shfl_down_sync(0xffffffffu, minV, c1);
			t1 = __shfl_down_sync(0xffffffffu, maxV, c1);
			if (t0 < minV)minV = t0;
			if (t1 > maxV)maxV = t1;
		}
		if (tid == 0)
		{
			minMax[idx] = minV;
			minMax[idx + 1] = maxV;
			constexpr unsigned int offsetSize(2 * blockXLv2 * blockYLv2);
			idx += offsetSize;
		}
		minV = v;
		maxV = v;
	}
}

__global__ void compatingLv2(float isoValue, float* minMax,
	unsigned int* blockIndicesLv1, unsigned int* blockIndicesLv2,
	unsigned int counterBlockNumLv1, unsigned int* countedBlockNumLv2)
{
	unsigned int laneid;
	asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
	constexpr unsigned int warpNum(countingThreadNumLv2 / 32);
	__shared__ unsigned int sums[32];
	unsigned int tid(threadIdx.x);
	unsigned int warpid(tid >> 5);
	unsigned int id0(tid + blockIdx.x * countingThreadNumLv2);
	unsigned int id1(id0 / voxelNumLv2);
	unsigned int test;
	if (id1 < counterBlockNumLv1)
	{
		if (minMax[2 * id0] <= isoValue && minMax[2 * id0 + 1] >= isoValue)test = 1;
		else test = 0;
	}
	else test = 0;
	unsigned int testSum(test);
#pragma unroll
	for (int c0(1); c0 < 32; c0 *= 2)
	{
		unsigned int tp(__shfl_up_sync(0xffffffffu, testSum, c0));
		if (laneid >= c0)testSum += tp;
	}
	if (laneid == 31)sums[warpid] = testSum;
	__syncthreads();
	if (warpid == 0)
	{
		unsigned warpSum = sums[laneid];
#pragma unroll
		for (int c0(1); c0 < warpNum; c0 *= 2)
		{
			unsigned int tp(__shfl_up_sync(0xffffffffu, warpSum, c0));
			if (laneid >= c0)warpSum += tp;
		}
		sums[laneid] = warpSum;
	}
	__syncthreads();
	if (warpid != 0)testSum += sums[warpid - 1];
	if (tid == countingThreadNumLv2 - 1)
		sums[31] = atomicAdd(countedBlockNumLv2, testSum);
	__syncthreads();
	if (test)
	{
		unsigned int bIdx1(blockIndicesLv1[id1]);
		unsigned int bIdx2;
		unsigned int x1, y1, z1;
		unsigned int x2, y2, z2;
		unsigned int tp1(bIdx1);
		unsigned int tp2((tid + blockIdx.x * countingThreadNumLv2) % voxelNumLv2);
		x1 = tp1 % gridXLv1;
		x2 = tp2 % blockXLv2;
		tp1 /= gridXLv1;
		tp2 /= blockXLv2;
		y1 = tp1 % gridYLv1;
		y2 = tp2 % blockYLv2;
		z1 = tp1 / gridYLv1;
		z2 = tp2 / blockYLv2;
		bIdx2 = x2 + blockXLv2 * (x1 + gridXLv1 * (y2 + blockYLv2 * (y1 + gridYLv1 * (z1 * blockZLv2 + z2))));
		blockIndicesLv2[testSum + sums[31] - 1] = bIdx2;
	}
}

__global__ void generatingTriangles(
	float isoValue, unsigned int* blockIndicesLv2,
	unsigned short const* distinctEdgesTable, int const* triTable, uchar4 const* edgeIDTable,
	unsigned int* countedVerticesNum, unsigned int* countedTrianglesNum, float* vertices, unsigned int* triangles)
{
	unsigned int blockId(blockIndicesLv2[blockIdx.x]);
	unsigned int tp(blockId);
	unsigned int x((tp % gridXLv2) * (voxelXLv2 - 1) + threadIdx.x);
	tp /= gridXLv2;
	unsigned int y((tp % gridYLv2) * (voxelYLv2 - 1) + threadIdx.y);
	unsigned int z((tp / gridYLv2) * (voxelZLv2 - 1) + threadIdx.z);
	__shared__ unsigned short vertexIndices[voxelZLv2][voxelYLv2][voxelXLv2];
	__shared__ float value[voxelZLv2 + 1][voxelYLv2 + 1][voxelXLv2 + 1];
	unsigned int eds(7);
	float v(value[threadIdx.z][threadIdx.y][threadIdx.x] = f(x, y, z));
	if (threadIdx.x == voxelXLv2 - 1)
	{
		eds &= 6;
		value[threadIdx.z][threadIdx.y][voxelXLv2] = f(x + 1, y, z);
		if (threadIdx.y == voxelYLv2 - 1)
			value[threadIdx.z][voxelYLv2][voxelXLv2] = f(x + 1, y + 1, z);
	}
	if (threadIdx.y == voxelYLv2 - 1)
	{
		eds &= 5;
		value[threadIdx.z][voxelYLv2][threadIdx.x] = f(x, y + 1, z);
		if (threadIdx.z == voxelZLv2 - 1)
			value[voxelZLv2][voxelYLv2][threadIdx.x] = f(x, y + 1, z + 1);
	}
	if (threadIdx.z == voxelZLv2 - 1)
	{
		eds &= 3;
		value[voxelZLv2][threadIdx.y][threadIdx.x] = f(x, y, z + 1);
		if (threadIdx.x == voxelXLv2 - 1)
			value[voxelZLv2][threadIdx.y][voxelXLv2] = f(x + 1, y, z + 1);
	}
	eds <<= 13;
	__syncthreads();
	unsigned int cubeCase(0);
	if (value[threadIdx.z][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 1;
	if (value[threadIdx.z][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 2;
	if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 4;
	if (value[threadIdx.z][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 8;
	if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x] < isoValue) cubeCase |= 16;
	if (value[threadIdx.z + 1][threadIdx.y][threadIdx.x + 1] < isoValue) cubeCase |= 32;
	if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x + 1] < isoValue) cubeCase |= 64;
	if (value[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x] < isoValue) cubeCase |= 128;
	unsigned int distinctEdges(eds ? distinctEdgesTable[cubeCase] : 0);
	unsigned int numTriangles(eds != 0xe000 ? 0 : distinctEdges & 7);
	unsigned int numVertices(__popc(distinctEdges &= eds));
	unsigned int laneid;
	asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
	unsigned warpid((threadIdx.x + voxelXLv2 * (threadIdx.y + voxelYLv2 * threadIdx.z)) >> 5);
	constexpr unsigned int threadNum(voxelXLv2 * voxelYLv2 * voxelZLv2);
	constexpr unsigned int warpNum(threadNum / 32);
	unsigned int sumVertices(numVertices);
	unsigned int sumTriangles(numTriangles);
	__shared__ unsigned int sumsVertices[32];
	__shared__ unsigned int sumsTriangles[32];
#pragma unroll
	for (int c0(1); c0 < 32; c0 *= 2)
	{
		unsigned int tp0(__shfl_up_sync(0xffffffffu, sumVertices, c0));
		unsigned int tp1(__shfl_up_sync(0xffffffffu, sumTriangles, c0));
		if (laneid >= c0)
		{
			sumVertices += tp0;
			sumTriangles += tp1;
		}
	}
	if (laneid == 31)
	{
		sumsVertices[warpid] = sumVertices;
		sumsTriangles[warpid] = sumTriangles;
	}
	__syncthreads();
	if (warpid == 0)
	{
		unsigned warpSumVertices = sumsVertices[laneid];
		unsigned warpSumTriangles = sumsTriangles[laneid];
#pragma unroll
		for (int c0(1); c0 < warpNum; c0 *= 2)
		{
			unsigned int tp0(__shfl_up_sync(0xffffffffu, warpSumVertices, c0));
			unsigned int tp1(__shfl_up_sync(0xffffffffu, warpSumTriangles, c0));
			if (laneid >= c0)
			{
				warpSumVertices += tp0;
				warpSumTriangles += tp1;
			}
		}
		sumsVertices[laneid] = warpSumVertices;
		sumsTriangles[laneid] = warpSumTriangles;
	}
	__syncthreads();
	if (warpid != 0)
	{
		sumVertices += sumsVertices[warpid - 1];
		sumTriangles += sumsTriangles[warpid - 1];
	}
	if (eds == 0)
	{
		sumsVertices[31] = atomicAdd(countedVerticesNum, sumVertices);
		sumsTriangles[31] = atomicAdd(countedTrianglesNum, sumTriangles);
	}
	/*__syncthreads();
	unsigned int interOffsetVertices(sumVertices - numVertices);
	sumVertices = interOffsetVertices + sumsVertices[31];//exclusive offset
	sumTriangles = sumTriangles + sumsTriangles[31] - numTriangles;//exclusive offset
	vertexIndices[threadIdx.z][threadIdx.y][threadIdx.x] = interOffsetVertices | distinctEdges;
	__syncthreads();
	for (unsigned int c0(0); c0 < numTriangles; ++c0)
	{
#pragma unroll
		for (unsigned int c1(0); c1 < 3; ++c1)
		{
			int edgeID(triTable[16 * cubeCase + 3 * c0 + c1]);
			uchar4 edgePos(edgeIDTable[edgeID]);
			unsigned short vertexIndex(vertexIndices[threadIdx.z + edgePos.z][threadIdx.y + edgePos.y][threadIdx.x + edgePos.x]);
			vertexIndex = __popc(vertexIndex >> (13 + edgePos.w)) + (vertexIndex & 0x1fff);
			triangles[3 * (sumTriangles + c0) + c1] = sumsVertices[31] + vertexIndex;
		}
	}
	if (distinctEdges & (1 << 15))
		vertices[sumVertices++] = zeroPoint(x, v, value[threadIdx.z][threadIdx.y][threadIdx.x + 1]);
	if (distinctEdges & (1 << 14))
		vertices[sumVertices++] = zeroPoint(y, v, value[threadIdx.z][threadIdx.y + 1][threadIdx.x]);
	if (distinctEdges & (1 << 13))
		vertices[sumVertices] = zeroPoint(z, v, value[threadIdx.z + 1][threadIdx.y][threadIdx.x]);*/
}

void convertToDistinctEdges()
{
	for (unsigned int c0(0); c0 < 256; ++c0)
	{
		unsigned int distinctEdges((edgeTable[c0] & 1) << 2);
		unsigned int n(0);
		while (triTable[c0][3 * n] >= 0)++n;
		distinctEdges |= (edgeTable[c0] >> 2) & (1 << 1);
		distinctEdges |= (edgeTable[c0] >> 8) & 1;
		distinctEdges <<= 13;
		distinctEdges |= n;
		::printf("0x%p,", distinctEdges);
		if ((c0 + 1 & 7) == 0)::printf("\n");
		else ::printf(" ");
	}
}

int main()
{
	std::uniform_real_distribution<float>rd(0, 1);
	std::mt19937 mt(time(nullptr));
	constexpr unsigned int threadNum(voxelXLv1 * voxelYLv1);
	constexpr size_t dataSize(threadNum * blockNum * sizeof(float));
	constexpr size_t minMaxSize(2 * blockNum * sizeof(float));
	//float* data((float*)::malloc(dataSize));
	//float* dataDebug((float*)::malloc(dataSize));
	//float* dataDevice;
	//float minV, maxV;
	//float* minMax((float*)::malloc(minMaxSize));
	//float* minMaxGPU((float*)::malloc(minMaxSize));
	float* minMaxLv1Device;
	float* minMaxLv2Device;
	unsigned int* blockIndicesLv1Device;
	unsigned int* blockIndicesLv2Device;
	unsigned int* countedBlockNumLv1Device;
	unsigned int* countedBlockNumLv2Device;
	unsigned short* distinctEdgesTableDevice;
	int* triTableDevice;
	uchar4* edgeIDTableDevice;
	unsigned int* countedVerticesNumDevice;
	unsigned int* countedTrianglesNumDevice;
	float* vertices;
	unsigned int* triangles;
	//hipMalloc(&dataDevice, dataSize);
	hipMalloc(&minMaxLv1Device, blockNum * 2 * sizeof(float));
	hipMalloc(&blockIndicesLv1Device, blockNum * sizeof(unsigned int));
	hipMalloc(&countedBlockNumLv1Device, sizeof(unsigned int));
	hipMalloc(&countedBlockNumLv2Device, sizeof(unsigned int));
	hipMalloc(&distinctEdgesTableDevice, sizeof(distinctEdgesTable));
	hipMalloc(&triTableDevice, sizeof(triTable));
	hipMalloc(&edgeIDTableDevice, sizeof(edgeIDTable));
	hipMalloc(&countedVerticesNumDevice, sizeof(unsigned int));
	hipMalloc(&countedTrianglesNumDevice, sizeof(unsigned int));
	hipMemcpy(distinctEdgesTableDevice, distinctEdgesTable, sizeof(distinctEdgesTable), hipMemcpyHostToDevice);
	hipMemcpy(triTableDevice, triTable, sizeof(triTable), hipMemcpyHostToDevice);
	hipMemcpy(edgeIDTableDevice, edgeIDTable, sizeof(edgeIDTable), hipMemcpyHostToDevice);
	//convertToDistinctEdges();
	for (unsigned int c0(0); c0 < 1; ++c0)
	{
		//for (unsigned int c1(0); c1 < blockNum; ++c1)
		//{
		//	minV = 1;
		//	maxV = 0;
		//	for (unsigned int c2(0); c2 < threadNum; ++c2)
		//	{
		//		float tp(rd(mt));
		//		data[c1 * threadNum + c2] = tp;
		//		if (tp < minV)minV = tp;
		//		if (tp > maxV)maxV = tp;
		//	}
		//	minMax[2 * c1] = minV;
		//	minMax[2 * c1 + 1] = maxV;
		//}
		//hipMemcpy(dataDevice, data, dataSize, hipMemcpyHostToDevice);
		Timer timer;
		hipDeviceSynchronize();
		timer.begin();
		float isoValue(-0.9f);
		unsigned int countedBlockNumLv1(0);
		unsigned int countedBlockNumLv2(0);
		unsigned int countedVerticesNum(0);
		unsigned int countedTrianglesNum(0);
		hipMemcpy(countedBlockNumLv1Device, &countedBlockNumLv1, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(countedBlockNumLv2Device, &countedBlockNumLv2, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(countedVerticesNumDevice, &countedVerticesNum, sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(countedTrianglesNumDevice, &countedTrianglesNum, sizeof(unsigned int), hipMemcpyHostToDevice);

		computeMinMaxLv1 << <GridSizeLv1, BlockSizeLv1 >> > (/*dataDevice, */minMaxLv1Device);
		compatingLv1 << <countingBlockNumLv1, countingThreadNumLv1 >> > (isoValue, minMaxLv1Device, blockIndicesLv1Device, countedBlockNumLv1Device);

		hipMemcpy(&countedBlockNumLv1, countedBlockNumLv1Device, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMalloc(&minMaxLv2Device, countedBlockNumLv1 * voxelNumLv2 * 2 * sizeof(float));

		computeMinMaxLv2 << < countedBlockNumLv1, BlockSizeLv2 >> > (blockIndicesLv1Device, minMaxLv2Device);

		hipMalloc(&blockIndicesLv2Device, countedBlockNumLv1 * voxelNumLv2 * sizeof(unsigned int));
		unsigned int countingBlockNumLv2((countedBlockNumLv1 * voxelNumLv2 + countingThreadNumLv2 - 1) / countingThreadNumLv2);

		compatingLv2 << <countingBlockNumLv2, countingThreadNumLv2 >> > (isoValue, minMaxLv2Device, blockIndicesLv1Device, blockIndicesLv2Device, countedBlockNumLv1, countedBlockNumLv2Device);

		hipMemcpy(&countedBlockNumLv2, countedBlockNumLv2Device, sizeof(unsigned int), hipMemcpyDeviceToHost);

		//hipMalloc(&vertices, countedBlockNumLv2 * 304 * sizeof(float));
		//hipMalloc(&triangles, countedBlockNumLv2 * 315 * 3 * sizeof(unsigned int));
		generatingTriangles << <countedBlockNumLv2, BlockSizeGenerating >> > (
			isoValue, blockIndicesLv2Device,
			distinctEdgesTableDevice, triTableDevice, edgeIDTableDevice,
			countedVerticesNumDevice, countedTrianglesNumDevice, vertices, triangles);
		hipDeviceSynchronize();
		timer.end();
		timer.print();
		::printf("Block Lv1: %u\nBlock Lv2: %u\n", countedBlockNumLv1, countedBlockNumLv2);
		::printf("Vertices Size: %u\n", countedBlockNumLv2 * 304 * sizeof(unsigned int));
		::printf("Triangles Size: %u\n", countedBlockNumLv2 * 315 * 3 * sizeof(unsigned int));
		hipMemcpy(&countedVerticesNum, countedVerticesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(&countedTrianglesNum, countedTrianglesNumDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
		::printf("Vertices: %u\nTriangles: %u\n", countedVerticesNum, countedTrianglesNum);
		unsigned int* debugBuffer((unsigned int*)::malloc(10 * sizeof(unsigned int)));
		hipMemcpy(debugBuffer, blockIndicesLv2Device, 10 * sizeof(unsigned int), hipMemcpyDeviceToHost);
		for (int c1(0); c1 < 10; ++c1)
			::printf("%u\n", debugBuffer[c1]);
		//::printf("%u %u\n", countedBlockNumLv1, countedBlockNumLv2);
		free(debugBuffer);
		hipFree(minMaxLv2Device);
		hipFree(blockIndicesLv2Device);
		//hipFree(vertices);
		//hipFree(triangles);

		//hipMemcpy(minMaxGPU, minMaxDevice, blockNum * 2 * sizeof(float), hipMemcpyDeviceToHost);
		//for (unsigned int c1(0); c1 < blockNum; ++c1)
		//	if (minMax[2 * c1] != minMaxGPU[2 * c1] || minMax[2 * c1 + 1] != minMaxGPU[2 * c1 + 1])
		//		::printf("%d\n", c1);
	}
	//hipFree(dataDevice);
	hipFree(minMaxLv1Device);
	hipFree(blockIndicesLv1Device);
	hipFree(countedBlockNumLv1Device);
	hipFree(countedBlockNumLv2Device);
	hipFree(distinctEdgesTableDevice);
	hipFree(triTableDevice);
	hipFree(edgeIDTableDevice);
	hipFree(countedVerticesNumDevice);
	hipFree(countedTrianglesNumDevice);
}